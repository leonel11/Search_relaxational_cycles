#include "hip/hip_runtime.h"
﻿#pragma region Формулировка задачи

	// Дан многомерный кубик. С помощью определенных входных параметров опредляются точки, из которых будут испускаться последовательности решений математического 
	// отображения. Требуется найти всевозможные цикловые решения без повторений, не превышающие определенной длины, в многомерном кубике.
	//
	// Вход: в файле data.config содержатся данные для работы программы
	//		dimension  (n, size) - размерность многомерного кубика;
	//		Далее идут dimension отрезков-проекций вида [a_i, b_i] многомерного куба на оси координат (полностью определяют расположение куба в пространстве) 
	//		(initset_proections);
	//		discr (d) - параметр дискретизации, расстояние между 2 соседними точками, из которых будут испускаться последовательности решений
	//					(параметр дискретизации определяет сетку (grid), которая накладывается на многомерный куб. Из каждой точки сетки испускаются решения);
	//		skip_iter (N, skip, amt_skip_iter) - количество пропускаемых значений для того, чтобы приблизиться к месту нахождения возможных циклов;
	//		max_iter (M, max_amt_iter) - ограничение сверху на количество итераций (максимальное количество подсчитанных значений в последовательности решений);
	//		max_cycle_len (K, lim_cycle_len) - максимально возможное значение длины цикла;
	//		T (check) - количество проверок цикла на устойчивость;
	//		eps - точность цикла (если какие-нибудь 2 значения в последовательности решений отличаются друг от друг не более чем на eps по модулю, 
	//			  то существует подозрение, что в этой последовательности решений есть цикл);
	//		tau - точность для проверки циклов на равенства (если 2 цикла одинаковой длины отличаются друг от друга по определенной выбранной норме не более чем на
	//			  tau по модулю, то эти циклы считаются равными);
	//		name_math_mapping - использумое математическое отображение;
	//		math_mapping_params - входные параметры математического изображения, НЕОБЯЗАТЕЛЬНЫЙ ПАРАМЕТРЫ (по умолчанию все входные параметры для математического
	//							  отображения - нулевые).
	//
	// Выход: в файле all_cycles.txt содержатся все возможные циклы в данном многомерном кубе;
	//		  в файле different_cycles.txt содержатся все различные циклы.

#pragma endregion 

#include "program.h" // вспомогательный заголовочный .h файл

int NumberMathMapping(char name_math_mapping[MAX_ARRAY_SIZE]) {
	string string_math_mapping(name_math_mapping, 0, strlen(name_math_mapping)-1);
	if (string_math_mapping == "ROTATION")
		return 1;
	if (string_math_mapping == "BERNULLI")
		return 2;
	if (string_math_mapping == "LOGICAL")
		return 3;
	if (string_math_mapping == "FILTERING")
		return 4;
	if (string_math_mapping == "RELAX")
		return 5;
	return 0;
}

int NumberGridKind(char name_grid[MAX_ARRAY_SIZE]) {
	string string_name_grid(name_grid, 0, strlen(name_grid)-1);
	if (string_name_grid == "UNIFORM")
		return 1;
	if (string_name_grid == "RANDOM")
		return 2;
	return 0;
}

bool AreParseDataFromConfig(string configfile, string readsection) { // распарсивание данных из файла конфигураций с помощью скрипта, написанного на Питоне
	FILE *script_file;
	char buffer[MAX_ARRAY_SIZE]; // выходящая информация из скрипта
	string callcommand; // строка вызова для функции _popen
	callcommand = "python " + pyscriptname + " " + configfile + " " + readsection; // формирование вызываемой в функции _popen строки
	if (!(script_file = _popen(callcommand.c_str(), "r"))) { // в том случае, если скрипт не выполнился
		printf("Error in launch Python script!");
		return false;
	}
	while (fgets(buffer, sizeof(buffer), script_file) != NULL) // выгрузка выходящей из скрипта информации о его работе
		printf("%s", buffer);
	_pclose(script_file);
	return true;
}

// функция считывания входных параметров для решения задачи
bool ReadData(string configfile, string readsection, int *dimension, float *a, float *b, int *num_points_source, float *discr, int *skip_iter, int *max_iter, int *max_cycle_len, int *T, float *eps, float *tau, int *num_math_mapping, float *math_mapping_params) { 
	if (!AreParseDataFromConfig(configfile, readsection))
		return false;
	FILE *input_file; // вспомогательный файл, с которого будет считываться информация
	input_file = fopen("input.txt", "rt");
	// считывание данных и проверка правильности считанных значений относительно ограничений
	fscanf(input_file, "%d", dimension);
	if (*dimension > MAX_DIMENSION) {
		printf("Too much dimension of the multidimensional cube! Please reduce it...\n");
		return false;
	}
	for (int i=0; i<MAX_DIMENSION; i++)
		a[i] = b[i] = 0.;
	for (int i = 0; i < *dimension; i++)
		fscanf(input_file, "%f %f", &a[i], &b[i]);
	char ch = NULL;
	fscanf(input_file, "%c", &ch);
	char name_points_source[MAX_ARRAY_SIZE];
	fgets(name_points_source, MAX_ARRAY_SIZE, input_file);
	*num_points_source = NumberGridKind(name_points_source);
	fscanf(input_file, "%f", discr);
	if (*discr > MAX_DISCR) {
		printf("Too high a value of the discretization! Please reduce it...\n");
		return false;
	}
	fscanf(input_file, "%d %d %d %d", skip_iter, max_iter , max_cycle_len, T);
	if (*max_cycle_len > LIM_MAX_CYCLE_LENGTH) {
		printf("Too high a value of cycle length! Please reduce it...\n");
		return false;
	}
	fscanf(input_file,"%f %f", eps, tau);
	memset(math_mapping_params, 0, MAX_ARRAY_SIZE);
	fscanf(input_file, "%c", &ch);
	char name_math_mapping[MAX_ARRAY_SIZE];
	fgets(name_math_mapping, MAX_ARRAY_SIZE, input_file);
	*num_math_mapping = NumberMathMapping(name_math_mapping);
	int idx = 0; 
	fscanf(input_file, "%d", &idx);
	for (int i=0; i<idx; i++)
		fscanf(input_file, "%f", &math_mapping_params[i]);
	printf("Program data is READ!\n"); // вывод сообщения об удачном считывании входных данных для работы программы
	fclose(input_file);
	remove("input.txt"); // удаление вспомогательного файла
	return true;
}

void OutputAllCycles(float a[MAX_ARRAY_SIZE], float discr, dim3 grid, int start_points, CycleData *cycle_list_host) { // вывод всех циклов в файл
	FILE *output_file; // файл для вывода информации
	output_file = fopen("all_cycles.txt", "wt");
	fprintf(output_file, "Start point \t\t\t\t Cyclelength \t\t\t Cycle\n");
	for (int i = 0; i < start_points; ++i) { // вывод координат точек, из которых испускались решения
		fprintf (output_file, "\n(%f,%f) \t\t cyclelength: %d \t\t", a[0]+(i%grid.x)*discr, a[1]+i/grid.y*discr, cycle_list_host[i].length);
		for (int j = 0; j < cycle_list_host[i].length; ++j) // вывод координат цикла
			fprintf (output_file, " (%f,%f)  ", cycle_list_host[i].points[j].x,  cycle_list_host[i].points[j].y) ;
	}
	fprintf (output_file, "\n\nAll start points: %d ", start_points) ;
	fclose(output_file);
	printf("Results of work program WRITE to output files!\n"); // вывод сообщения об удачном выводе данных в файл
}

void CudaOutputErrors() { // функция, выводящая информацию об ошибках при работе с инструментом CUDA
	printf("CUDA errors: ");
	printf(hipGetErrorString(hipGetLastError()));
	printf("\n\n");
}

void ResultsOfWorkProgram(bool state) { // вывод информации о завершающих этапах работы программы
	if (!state)
		printf("Program is interrupted!\n");
	else {
		printf("Work with GPU are OVER!\nProgram has successfully completed its work!\n");
		CudaOutputErrors();
	}
}

bool IsCudaDataCreated(void *cuda_pointer) { // функция, проверяющая была ли выделена память на GPU
	if (cuda_pointer == NULL) { // если память под указатель не была выделена на GPU
		printf("\nData for device is not created!\n");
		CudaOutputErrors();
		return 0;
	}
	return 1;
}

// функция очистки памяти от динамических объектов
void FreeUsedDynamicObjects(CycleData *cycle_list_host, CycleData *cycle_list_device, float *a_device, float *math_mapping_params, float2 *point_seq_device) { 
	free(cycle_list_host);
	hipFree(cycle_list_device);
	hipFree(a_device);
	hipFree(math_mapping_params);
	hipFree(point_seq_device);
}

float RandomBetweenTwoValues(float p, float q) {
	float rand_num = ((float)rand()) / (float)RAND_MAX;
	float diff = q - p;
	float r = rand_num * diff;
	return p + r;
}

bool MakeGrid(int num_points_source, dim3 *grid, float a[MAX_ARRAY_SIZE], float b[MAX_ARRAY_SIZE], float discr) { // формирование сетки для GPU
	if (num_points_source == 0) { // не был определён способ формирования сетки с начальными точками
		printf("\nGrid was not formed!\n");
		CudaOutputErrors();
		return false;
	}
	if (num_points_source == 1) {
		grid->x = ((b[0] - a[0]) / discr) + 1;
		grid->y = ((b[1] - a[1]) / discr) + 1;
	}
	if (num_points_source == 2) {
		grid->x = RandomBetweenTwoValues(a[0], b[0]);
		grid->y = RandomBetweenTwoValues(a[1], b[1]);
	}
	return true;
}

__device__ void SkipIterations(float2 *point, int num_iter, int skip_iter, int num_math_mapping, float math_mapping_params[MAX_ARRAY_SIZE]) { // пропуск первых N итераций
	for (int i = 0; i < skip_iter; ++i)
		*point = MathMapping(*point, num_iter+i, num_math_mapping, math_mapping_params);
}

__device__ bool IsEqualPoints(float2 first_point, float2 second_point, float eps) { // эквивалентные точки с точностью до eps
	if ((fabs(first_point.x - second_point.x) < eps) && (fabs(first_point.y - second_point.y) < eps))
		return 1;
	return 0;
}

float hostEuclideanDifference(float2 first_point, float2 second_point) { // разница между 2 циклами, посчитанная с помощью второй (евклидовой) нормы
	return sqrt(powf(fabs(first_point.x - second_point.x), 2) + powf((fabs(first_point.y - second_point.y)), 2));
}

int GetAmountAllCycles(int start_points, CycleData *cycle_list_host) {
	int cycles = 0; // количество циклов
	for (int i = 0; i < start_points; i++) {
		if (cycle_list_host[i].length > 0)
			cycles++;
	}
	return cycles;
}

// формирование списка различных циклов
void FormDifferentCyclesList(float *a, float discr, dim3 grid, int start_points, vector <DifferingCycleInfo> *differentCycles, CycleData *cycle_list_host, float tau) {
	bool is_add_cycle = false;
	for (int i = 0; i < start_points; ++i) {
		is_add_cycle = false;
		for (int j = 0; j < differentCycles->size(); ++j) {
			if (cycle_list_host[i].length == 0) { // циклы длины 0 не добавляются в вектор различных циклов
				is_add_cycle = true;
				break;
			}
			// добавление циклов в зависимости от значения евклидовой нормы разности точек
			int h = 0;
			for ( ; h < (*differentCycles)[j].elements.size(); ++h) 
				if (hostEuclideanDifference((*differentCycles)[j].elements[h], cycle_list_host[i].points[h]) >= tau)
					break;
			if (h == cycle_list_host[i].length) {
				(*differentCycles)[j].startpoints.push_back(make_float2(a[0]+(i%grid.x)*discr, a[1]+i/grid.y*discr));
				is_add_cycle = true;
				break;
			}
		}
		if (!is_add_cycle && cycle_list_host[i].length) { // добавление цикла в вектор
			DifferingCycleInfo ci;
			for (int j = 0; j < cycle_list_host[i].length; ++j)
				ci.elements.push_back(cycle_list_host[i].points[j]);
			ci.startpoints.push_back(make_float2(a[0]+(i%grid.x)*discr, a[1]+i/grid.y*discr));
			differentCycles->push_back(ci);
		}
	}
}

void OutputDifferentCycles(float *a, float d, dim3 grid, int start_points, CycleData *cycle_list_host, float tau) { // вывод различных циклов в файл
	vector <DifferingCycleInfo> differentCycles; // вектор, содержащий различные циклы
	int cycles = GetAmountAllCycles(start_points, cycle_list_host);
	FormDifferentCyclesList(a, d, grid, start_points, &differentCycles, cycle_list_host, tau);
	// вывод различных циклов
	FILE *output_file; // файл для вывода информации
	output_file = fopen("different_cycles.txt", "wt");
	fprintf(output_file, "All start points: \t%d\n", start_points);
	fprintf(output_file, "All cycles: \t\t%d\n", cycles);
	fprintf(output_file, "\nDifferent cycles \t%d\t:\n",  differentCycles.size());
	for (int i = 0; i < differentCycles.size(); ++i) {
		fprintf(output_file, "\n------------------------------------------------");
		fprintf(output_file, "\nStart points \t%d\t:\t", differentCycles[i].startpoints.size());
		for (int j = 0; j < differentCycles[i].startpoints.size(); ++j)
			fprintf(output_file, "(%f, %f)\t", differentCycles[i].startpoints[j].x, differentCycles[i].startpoints[j].y);
		fprintf(output_file, "\nCycle elements \t%d\t:\t", differentCycles[i].elements.size());
		for (int j = 0; j < differentCycles[i].elements.size(); ++j)
			fprintf(output_file, "(%f, %f)\t", (differentCycles[i].elements[j]).x, differentCycles[i].elements[j].y);	
	}
	fclose(output_file);
}

bool IsPointSequenceOver(float2 point) { // проверка на окончание последовательности-решения
	float2 stop_point = make_float2(FLT_MAX, FLT_MAX); // точка с очень большими координатами, символизирующими конец последовательности-решения
	if (point.x != stop_point.x || point.y != stop_point.y )
		return false;
	else
		return true;
}

void OutputAllPointSequences(float2 point_seq[MAX_ARRAY_SIZE], int start_points, int max_cycle_len) { // вывод всех последовательностей-решений в файл
	FILE *output_file;
	output_file = fopen("all_pointsequences.txt", "w");
	fprintf(output_file, "Start point \t\t\t\t Points of the sequence\n");
	for (int i = 0; i < start_points; ++i) {
		fprintf(output_file, "\n(%f, %f) \t\t", point_seq[i*max_cycle_len].x, point_seq[i*max_cycle_len].y);
		for (int j = 1; j < max_cycle_len; ++j) {
			if (!IsPointSequenceOver(point_seq[i*max_cycle_len+j]))
				fprintf(output_file, "(%f, %f) ", point_seq[i*max_cycle_len+j].x, point_seq[i*max_cycle_len+j].y);
			else
				break;
		}
	}
	fclose(output_file);
}

// избавление от значений вида -0.000000 при выводе в файл (подобного рода значения возникают из-за машинного нуля)
void DeleteNegativeZeros(int start_points, CycleData *cycle_list_host) {
	for (int i = 0; i < start_points; ++i)
		for (int j = 0; j < cycle_list_host[i].length; ++j) {
			if (cycle_list_host[i].points[j].x == 0)
				cycle_list_host[i].points[j].x = 0;
			if (cycle_list_host[i].points[j].y == 0)
				cycle_list_host[i].points[j].y = 0;
		}
}

__device__ float deviceEuclideanDifference(float2 first_point, float2 second_point) { // аналог функции hostEuclideanDifference, выполняется на GPU
	return __fsqrt_rn(__powf(fabs(first_point.x-second_point.x), 2) + __powf(fabs(first_point.y-second_point.y), 2));
}

// Проверка цикла на "устойчивость"
__device__ bool IsCyclePersisting(float2 first_point, int num_iter, int cycle_len, int T, float tau, int num_math_mapping, float math_mapping_params[MAX_ARRAY_SIZE]) { 
	float2 second_point = first_point;
	SkipIterations(&second_point, num_iter, cycle_len, num_math_mapping, math_mapping_params);
	num_iter+=cycle_len;
	for (int i = 0; i < T*cycle_len; ++i) {
		float diff = deviceEuclideanDifference(first_point, second_point);
		if (diff > tau)
			return 0;
		first_point = MathMapping(first_point, num_iter-cycle_len, num_math_mapping, math_mapping_params);
		second_point = MathMapping(second_point, num_iter, num_math_mapping, math_mapping_params);
		num_iter++;
	}
	return 1;
}

__device__ void AddCycleAtList(int idx, int cycle_len, float2 first_point, int num_iter, CycleData *cycle_list, int num_math_mapping, float math_mapping_params[MAX_ARRAY_SIZE]) { 
	// добавить в список по индексу idx k-элементов, начиная с first_point
	for (int i = 0; i < cycle_len; ++i) {
		cycle_list[idx].points[i] = first_point;
		first_point = MathMapping(first_point, num_iter, num_math_mapping, math_mapping_params);
		num_iter++;
	}
	cycle_list[idx].length = cycle_len;
}

__device__ void SearchMinPointInCycle(int idx, float2 *min, int *minPointIdx, CycleData *cycle_list) { // поиск минимальной точки в цикле и ее индекса
	for (int i = 0; i < cycle_list[idx].length ; ++i){
		if (cycle_list[idx].points[i].x < min->x) {
			*minPointIdx = i;
			*min = cycle_list[idx].points[i];
		}
		if (cycle_list[idx].points[i].x == min->x && cycle_list[idx].points[i].y < min->y) {
			*minPointIdx = i;
			*min = cycle_list[idx].points[i];
		}
	}
}

// циклический поворот (сдвиг) цикла относительно его минимального элемента
__device__ void TurnCycleRelativeHisMinElement(CycleData *cycle_list, int idx, int minPointIdx) { 
	float2 v[128];
	for (int i = 0; i < minPointIdx; ++i)
		v[i] = cycle_list[idx].points[i];
	for (int i = 0; i < cycle_list[idx].length-minPointIdx; ++i )
		cycle_list[idx].points[i] = cycle_list[idx].points[minPointIdx+i];
	for (int i = cycle_list[idx].length-minPointIdx; i < cycle_list[idx].length; ++i)
		cycle_list[idx].points[i] = v[i-cycle_list[idx].length+minPointIdx];
}

__global__ void KernelShiftCycle(CycleData * cycle_list) { // ядро, осуществляющее смещение (поворот) цикла
	int idx = blockIdx.x + blockIdx.y*gridDim.x;
	float2 min;
	if (cycle_list[idx].length != 0)
		min = cycle_list[idx].points[0];
	else
		return;
	int minPointIdx = 0; // индекс (номер) минимального элемента в цикле
	SearchMinPointInCycle(idx, &min, &minPointIdx, cycle_list);
	TurnCycleRelativeHisMinElement(cycle_list, idx, minPointIdx);
}

// ядро, осуществляющее поиск циклов
__global__ void KernelSearchCycles(float2 *point_seq, float a[MAX_ARRAY_SIZE], float discr, int skip_iter, int max_cycle_len, int T, CycleData *cycle_list, float eps, float tau, int num_math_mapping, float math_mapping_params[MAX_ARRAY_SIZE]) {
	float x = a[0] + blockIdx.x*discr; // x-координата текущей стартовой точки (из начального множества) 
	float y = a[1] + blockIdx.y*discr; // y-координата	
	int idx = blockIdx.x + blockIdx.y*gridDim.x;
	float2 first_point, second_point;
	first_point = make_float2(x, y);
	point_seq[idx*max_cycle_len+0]=first_point;
	SkipIterations(&first_point, 0, skip_iter, num_math_mapping, math_mapping_params); 
	int num_iter=skip_iter+1;
	second_point = MathMapping(first_point, skip_iter, num_math_mapping, math_mapping_params); // следующая точка из последовательности-решения
	point_seq[idx*max_cycle_len+1]=second_point;
	if( num_math_mapping==5 )
		max_cycle_len=math_mapping_params[4]+1;
	for (int i = 0; i < max_cycle_len-2; ++i) {
		if (IsEqualPoints(first_point, second_point, eps)) // сработало условие - значит first и second близки 
			if (IsCyclePersisting(first_point, skip_iter, i+1, T, tau, num_math_mapping, math_mapping_params)) { // если цикл сохраняется - добавляем элементы в список и выходим.
				if( num_math_mapping==5 )
					AddCycleAtList(idx, 1, first_point, skip_iter, cycle_list, num_math_mapping, math_mapping_params);
				else
					AddCycleAtList(idx, i+1, first_point, skip_iter, cycle_list, num_math_mapping, math_mapping_params);
				// поскольку в CUDA невозможно ни хранить зубчатые массивы, ни работать с vector, для окончания последовательности-решения, исходящего из данной 
				// начальной точки в её конец специально добавляется точка с очень большими координатами, символизирующее окончание работы с данной 
				// последовательностью-решением
				point_seq[idx*max_cycle_len+2+i]=make_float2(FLT_MAX,FLT_MAX);
				return;
			} // не сработал return, значит ищем цикл большей длины.
		if( num_math_mapping==5 )
			first_point=second_point;
		second_point = MathMapping(second_point, num_iter+i, num_math_mapping, math_mapping_params);
		point_seq[idx*max_cycle_len+2+i]=second_point;
	}
	cycle_list[idx].length = 0; // в случае если цикл так и не нашли
}

int main() {
	printf("Stages of program:\n\n"); // этапы выполнения программы
	int num_math_mapping = 0; // число, соответствующее математическому отображению
	int num_points_source = 0; // число, соответствующее разновидности генерации сетки
	int dimension = 0, skip_iter = 0, max_iter = 0, max_cycle_len = 0, T = 0;
	float eps = 0.0, tau = 0.0, discr = 0.0;
	float a[MAX_ARRAY_SIZE], b[MAX_ARRAY_SIZE]; // массивы для хранения концов отрезков-проекций многомерного куба на оси координат
	float math_mapping_params[MAX_ARRAY_SIZE]; // массив для хранения параметров математического отображения
	float2 point_seq[MAX_ARRAY_SIZE]; // массив для хранения последовательности-решения для каждой итерации
	string configfile = "data.config"; // конфигурационный файл, в котором хранятся входные параметры программы
	string readsection = "data_program"; // секция, определяющая какие данные следует считывать из конфига
	if (!ReadData(configfile, readsection, &dimension, a, b, &num_points_source, &discr, &skip_iter, &max_iter, &max_cycle_len, &T, &eps, &tau, &num_math_mapping, math_mapping_params)) { // в случае ошибки при считывании входных параметров программы
		ResultsOfWorkProgram(false);
		return 0;
	}
	dim3 grid; // сетка из начальных точек, поступающих на вход ядру
	if (!MakeGrid(num_points_source, &grid, a, b, discr)) {
		ResultsOfWorkProgram(false);
		return 0;
	}
	int start_points = grid.x*grid.y; // количество начальных точек, из которых будут испускаться решения
	CycleData *cycle_list_device = NULL, *cycle_list_host = NULL;
	float2 *point_seq_device = NULL;
	cycle_list_host = (CycleData*) malloc (start_points*sizeof(CycleData));
	hipMalloc(&cycle_list_device, start_points*sizeof(CycleData));
	if (!IsCudaDataCreated((void*) cycle_list_device))
		return 0;
	float *a_device = NULL, *math_mapping_params_device = NULL;
	hipMalloc(&a_device, dimension*sizeof(float));
	hipMalloc(&math_mapping_params_device, MAX_ARRAY_SIZE*sizeof(float));
	hipMalloc( &point_seq_device, MAX_ARRAY_SIZE*sizeof(float2));
	if (!IsCudaDataCreated((void*) a_device))
		return 0;
	if (!IsCudaDataCreated((void*) math_mapping_params_device))
		return 0;
	hipEvent_t start_time, finish_time;
	hipEventCreate(&start_time);
	hipEventCreate(&finish_time);
	hipEventRecord(start_time, 0);
	hipMemcpy(a_device, a, dimension*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(math_mapping_params_device, math_mapping_params, MAX_ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	KernelSearchCycles<<<grid, 1>>>(point_seq_device, a_device, discr, skip_iter, max_cycle_len, T, cycle_list_device, eps, tau, num_math_mapping, math_mapping_params_device);
	hipDeviceSynchronize(); // синхронизация всех нитей (следующая функция программы не будет выполнена до тех пор, пока последняя нить не окончит свою работу)
	KernelShiftCycle<<<grid, 1>>>(cycle_list_device);
	hipDeviceSynchronize();
	hipMemcpy(cycle_list_host, cycle_list_device, start_points*sizeof(CycleData), hipMemcpyDeviceToHost);
	hipMemcpy(point_seq, point_seq_device, MAX_ARRAY_SIZE*sizeof(float2), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(finish_time, 0);
	hipEventSynchronize(finish_time);
	float work_time = 0.0;
	hipEventElapsedTime(&work_time, start_time, finish_time);
	printf("Work time on GPU: %4.0f ms\n", work_time); // вывод времени работы программы на GPU 
	hipEventDestroy(start_time);
	hipEventDestroy(finish_time);
	OutputAllPointSequences(point_seq, start_points, max_cycle_len);
	DeleteNegativeZeros(start_points, cycle_list_host);
	OutputAllCycles(a, discr, grid, start_points, cycle_list_host);
	OutputDifferentCycles(a, discr, grid, start_points, cycle_list_host, tau);
	ResultsOfWorkProgram(true);
	FreeUsedDynamicObjects(cycle_list_host, cycle_list_device, a_device, math_mapping_params_device, point_seq_device);	
	return 0;
}